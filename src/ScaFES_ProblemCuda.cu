
#include <hip/hip_runtime.h>
#include <vector>

/*******************************************************************************
*******************************************************************************/
__global__ void kernelUpdateInnerVect(
    short int* const kindElemData, std::vector<double*> dfDepElemData,
    std::vector<double*> const dfIndepElemData, int const timeIter,
    std::vector<int> const setGridNodes, int const nNodesTotalPartition)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int ii = col * nNodesTotalPartition + row;
    // TODO: Index globNode = ...(ii);

    if (ii < nNodesTotalPartition)
    {
        // Call of OwnProblem::updateInner(...);
    }
}

/*******************************************************************************
*******************************************************************************/
int host_kernelUpdateInnerVect(
           dim3 blocksPerGrid,
           dim3 threadsPerBlock,
    short int* const kindElemData, std::vector<double*> dfDepElemData,
    std::vector<double*> const dfIndepElemData, int const timeIter,
    std::vector<int> const setGridNodes, int const nNodesTotalPartition)
{
    kernelUpdateInnerVect<<<blocksPerGrid,threadsPerBlock>>>(kindElemData,
                                                             dfDepElemData,
                                                             dfIndepElemData,
                                                             timeIter,
                                                             setGridNodes,
                                                             nNodesTotalPartition);
    return EXIT_SUCCESS;
}

// /*******************************************************************************
// *******************************************************************************/
// template<class OWNPRBLM, std::size_t DIM, typename CT>
// __global__ void kernelUpdateInnerCuda(DataField<short
// int,DIM> const& mKind,
//                                   std::vector< DataField<CT,DIM> >& dfDep,
//                                   std::vector< DataField<CT,DIM> > const& dfIndep,
//                                   int const& timeIter,
//                                   std::vector<int> const& setGridNodes,
//                                   int const& nNodesTotalPartition)
// {
//    int col = blockIdx.x * blockDim.x + threadIdx.x;
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int ii = col * nNodesTotalPartition + row;
//
//    if (ii < nNodesTotalPartition) {
//      Ntuple<int, DIM> idxNode = mKind.memoryPos2IdxNode(ii);
//      static_cast<OWNPRBLM*>(this)->template updateInner<CT>(dfDep, dfIndep,
//                                                             idxNode, timeIter);
//    }
// }
//
// /*******************************************************************************
// *******************************************************************************/
// template<class OWNPRBLM, std::size_t DIM>
// template<typename CT>
// inline void Problem<OWNPRBLM,
// DIM>::updateGlobalInnerUsingCuda(DataField<short int,DIM> const& mKind,
//                                   std::vector< DataField<CT,DIM> >& dfDep,
//                                   std::vector< DataField<CT,DIM> > const&
// dfIndep,
//                                   int const& timeIter,
//                                   std::vector<int> const& setGridNodes,
//                                   int const& nNodesTotalPartition)
// {
//    /*---------------------------------------------------------------------------
//    | dimBlock = number of threads per block
//    | dimGrid = number of blocks
//    ---------------------------------------------------------------------------*/
//    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
//    dim3 dimGrid(ceil((nNodesTotalPartition + dimBlock.x - 1) / dimBlock.x),
//                 ceil((nNodesTotalPartition + dimBlock.y - 1) / dimBlock.y));
//
//    kernelUpdateInner<<<dimGrid,dimBlock>>>(kind, dfDep, dfIndep,
//                                            timeIter,
//                                            nodeType,
//                                            nNodesTotalPartition);
//    cudaThreadSynchronize();
//    check_cuda_errors(__FILE__, __LINE__);
// }
//
